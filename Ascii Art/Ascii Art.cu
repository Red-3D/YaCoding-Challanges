#include "hip/hip_runtime.h"
#include<vector>
#include<fstream>
#include<stdio.h>
#include<iostream>
#include<filesystem>
#include"Reds_var_defs.hpp"

#include<hip/hip_runtime.h>
#include<>

#define fread(a, size) read((char*)&a, size)
#define threads 256

__global__ void process_image(u64 size, u16 char_amount, char* chars, float* density_map, u8 *img, char *output) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < size) {
		//red, green, blue, average
		u8 r, g, b, a;
		r = img[tid * 3];
		g = img[tid * 3 + 1];
		b = img[tid * 3 + 2];
		a = (r+g+b)/3;

		float min = INFINITY;
		char out;

		float density = (float)a/255;
		for (uint i = 0; i < char_amount; i++) {
			if (abs(density - density_map[chars[i] - 32]) < min) {
				min = abs(density - density_map[chars[i] - 32]);
				out = chars[i];
			}
		}
		output[tid] = out;
	}
	return;
}

int main() {

	//	/-------------\
	//	| device info |
	//	\-------------/
	int dev_ammount;
	hipDeviceProp_t dev;
	std::string uuid;
	hipGetDeviceCount(&dev_ammount);

	std::cout << "----------------------------------------------\n";
	for (int i = 0; i < dev_ammount; i++) {
		hipGetDeviceProperties(&dev, i);
		std::cout << "id:" << i << '\n';
		std::cout << " - name:         " << dev.name << '\n';
		std::cout << " - gpu clock:    " << dev.clockRate / 1000 << "mhz\n";
		std::cout << " - memory clock: " << dev.memoryClockRate / 1000 << "mhz\n";
		std::cout << " - Compute:      " << dev.major << '.' << dev.minor << '\n';
		std::cout << " - tpb           " << dev.maxThreadsPerBlock << '\n';
	}
	std::cout << "----------------------------------------------\n";

	//	/-----------------\
	//	| filestream shit |
	//	\-----------------/

	std::string path = "D:\\CPP\\C++ Projects\\YaCoding\\Ascii_Art\\images\\HD_Bobby.ppm";


	std::ifstream file(path, std::ios::in | std::ios::binary);
	if (!file.is_open()) {
		std::cout << "ERR[101] could not open: " << path.c_str() << "\n\n";
		exit(101);
	}
	char read;

	//	/-----------\
	//	| read file |
	//	\-----------/
	//magic number
	char identifier[2];
	file.fread(identifier, 2);
	if ((identifier[0] != 'P') | (identifier[1] != '6')) {
		std::cout << "ERR[102] this aint a ppm, identifier is: " << identifier[0] << identifier[1] << "\n\n";
		file.close();
		exit(102);
	}
	else {
		std::cout << "\n loaded [" << path << "], processing...\n\n";
	}

	//whitespace
	file.fread(read, 1);
	read = '0';

	//width
	u16 width = 0;
	file.fread(read, 1);
	while ((read >= '0') && (read <= '9')) {
		width *= 10;
		width += atoi(&read);
		file.fread(read, 1);
	}
	std::cout << " ----------------\n | width  | " << width << " |\n ----------------\n";
	read = '0';

	//height
	u16 height = 0;
	file.fread(read, 1);
	while ((read >= '0') && (read <= '9')) {
		height *= 10;
		height += atoi(&read);
		file.fread(read, 1);
	}
	std::cout << " | height | " << height << " |\n ----------------\n";
	read = '0';

	//maxcol
	u16 maxcol = 0;
	file.fread(read, 1);
	while ((read >= '0') && (read <= '9')) {
		maxcol *= 10;
		maxcol += atoi(&read);
		file.fread(read, 1);
	}
	if (maxcol != 255) {
		std::cout << "\nERR[103] maxcol is not 255, image not supported.\n\n";
		file.close();
		exit(103);
	}
	std::cout << " | maxcol | " << maxcol << " |\n ----------------\n\n";
	read = '0';

	//pixels
	u8* image = new u8[height * width * 3];
	file.read((char*)image, (height * width) * 3);

	//	/----------------------------------------------------------\
	//	|                      Character set                       |
	//	| duck users and their need to controll stuff, smh my head |
	//	\----------------------------------------------------------/
	char* charset;
	u16 char_amount = 0;
	char input = 0;
	std::cout << "choose a charset:\n - [0]: Custom\n - [1]: [^:.-=+\'\"@\\/]\n - [2]: [*~,#]\n";
ccs:
	input = getchar();
	switch (input) {
	case '1': {
		charset = "^:.-=+'\"@\\/";
		char_amount = 11;
		break;
	}
	case '2': {
		charset = "*~,#";
		char_amount = 4;
		break;
	}
	case '0': {
		std::cout << "\n - input charset: ";
		std::string tmp;
		std::cin >> tmp;
		charset = new char[tmp.length() + 1];
		strcpy(charset, tmp.c_str());
		char_amount = tmp.length();
		break;
	}
	default: {
		input = 0;
		std::cin.clear();
		goto ccs;
		break;
	}
	}
	std::cout << "\nusing charset: [" << charset << "] | length: " << char_amount << '\n';


	//	/------------\
	//	| Cuda Stuff |
	//	\------------/
	//host stuff
	char* host_out = new char[height * width];
	float density[95] = { 0.0f, 0.292538f, 0.244776f, 0.665671f, 0.752239f, 0.770149f, 0.752239f, 0.122388f, 0.349255f, 0.355223f, 0.331342f, 0.313432f, 0.18209f, 0.107463f, 0.101493f, 0.310448f, 0.674626f, 0.489553f, 0.525373f, 0.438805f, 0.549254f, 0.540299f, 0.573135f, 0.405969f, 0.695523f, 0.549254f, 0.191045f, 0.277612f, 0.280596f, 0.304477f, 0.283583f, 0.361194f, 1.0f, 0.614925f, 0.662687f, 0.468656f, 0.677613f, 0.555225f, 0.420895f, 0.629851f, 0.573135f, 0.447762f, 0.385075f, 0.537312f, 0.340297f, 0.704478f, 0.725372f, 0.58806f, 0.555225f, 0.808956f, 0.650745f, 0.540299f, 0.37612f, 0.519402f, 0.552238f, 0.728359f, 0.602984f, 0.432836f, 0.534328f, 0.42985f, 0.310448f, 0.42985f, 0.197015f, 0.197015f, 0.0597015f, 0.537312f, 0.552238f, 0.337313f, 0.570148f, 0.504476f, 0.450746f, 0.805969f, 0.498508f, 0.41194f, 0.480598f, 0.489553f, 0.420895f, 0.629851f, 0.432836f, 0.462685f, 0.56418f, 0.56418f, 0.340297f, 0.41791f, 0.432836f, 0.432836f, 0.405969f, 0.543283f, 0.459701f, 0.519402f, 0.423881f, 0.426865f, 0.35821f, 0.42985f, 0.220895f };

	//device stuff
	u8* dev_image;
	char* dev_out;
	char* dev_charset;
	float* dev_density;

	hipMalloc(&dev_image, 3 * (height * width));
	hipMemcpy(dev_image, image, 3 * (height * width), hipMemcpyHostToDevice);
	hipMalloc(&dev_out, height * width);
	hipMalloc(&dev_charset, char_amount);
	hipMemcpy(dev_charset, charset, char_amount, hipMemcpyHostToDevice);
	hipMalloc(&dev_density, sizeof(density));
	hipMemcpy(dev_density, density, sizeof(density), hipMemcpyHostToDevice);

	//launch kernels, sync and copy back to host
	process_image<<<(height * width) / threads + 1, threads>>>(height * width, char_amount, dev_charset, dev_density, dev_image, dev_out);
	hipDeviceSynchronize();
	hipMemcpy(host_out, dev_out, height * width, hipMemcpyDeviceToHost);
	std::cout << "\n ------------------\n | cuda |  done   |\n ------------------\n";

	//	/--------\
	//	| output |
	//	\--------/
	std::ofstream file_out("out.txt", std::ios::out);
	std::string tmp;

	for (u64 i = 0; i < height * width; i++) {
		tmp += host_out[i];
		tmp += ' ';
		if ((i + 1) % width == 0) {
			tmp += '\n';
		}
	}

	file_out << tmp;
	file_out.close();

	std::cout << " | file | written |\n ------------------\n";

	//	/----------------------------------------------\
	//	| Be a responsable dev and recycle your memory |
	//	\----------------------------------------------/
	hipFree(dev_image);
	hipFree(dev_out);
	hipFree(dev_charset);
	hipFree(dev_density);
	delete[] image;
	delete[] host_out;
	file.close();
	return 0;
}